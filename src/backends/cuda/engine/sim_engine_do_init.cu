#include "hip/hip_runtime.h"
#include <animator/global_animator.h>
#include <collision_detection/global_trajectory_filter.h>
#include <contact_system/global_contact_manager.h>
#include <diff_sim/global_diff_sim_manager.h>
#include <dof_predictor.h>
#include <fstream>
#include <global_geometry/global_simplicial_surface_manager.h>
#include <global_geometry/global_vertex_manager.h>
#include <gradient_hessian_computer.h>
#include <line_search/line_searcher.h>
#include <linear_system/global_linear_system.h>
#include <sim_engine.h>
#include <uipc/common/log.h>

namespace uipc::backend::cuda
{
void SimEngine::build()
{
    // 1) build all systems
    build_systems();

    // 2) find those engine-aware topo systems
    m_global_vertex_manager     = &require<GlobalVertexManager>();
    m_dof_predictor             = &require<DofPredictor>();
    m_line_searcher             = &require<LineSearcher>();
    m_gradient_hessian_computer = &require<GradientHessianComputer>();
    m_global_linear_system      = &require<GlobalLinearSystem>();

    m_global_simplicial_surface_manager = find<GlobalSimpicialSurfaceManager>();
    m_global_contact_manager            = find<GlobalContactManager>();
    m_global_trajectory_filter          = find<GlobalTrajectoryFilter>();
    m_global_animator                   = find<GlobalAnimator>();
    m_global_diff_sim_manager           = find<GlobalDiffSimManager>();

    // 3) dump system info
    dump_system_info();
}

void SimEngine::init_scene()
{
    auto& info            = world().scene().info();
    m_newton_velocity_tol = info["newton"]["velocity_tol"];
    m_newton_max_iter     = info["newton"]["max_iter"];
    m_ccd_tol             = info["newton"]["ccd_tol"];
    m_friction_enabled    = info["contact"]["friction"]["enable"];
    m_strict_mode         = info["extras"]["strict_mode"]["enable"];
    Vector3 gravity       = info["gravity"];
    Float   dt            = info["dt"];

    m_abs_tol = m_newton_velocity_tol * dt;

    event_init_scene();

    // some systems should be initialized after the scene is built
    m_global_linear_system->init();
    m_global_vertex_manager->init();
    m_global_simplicial_surface_manager->init();
    if(m_global_contact_manager)
        m_global_contact_manager->init();
    if(m_global_animator)
        m_global_animator->init();
    if(m_global_diff_sim_manager)
        m_global_diff_sim_manager->init();
}

void SimEngine::do_init(InitInfo& info)
{
    try
    {
        // 1. Build all the systems and their dependencies
        m_state = SimEngineState::BuildSystems;
        build();

        // 2. Trigger the init_scene event, systems register their actions will be called here
        m_state = SimEngineState::InitScene;
        init_scene();

        // 3. Any creation and deletion of objects after this point will be pending
        world().scene().begin_pending();
    }
    catch(const SimEngineException& e)
    {
        spdlog::error("SimEngine init error: {}", e.what());
        status().push_back(core::EngineStatus::error(e.what()));
    }
}
}  // namespace uipc::backend::cuda